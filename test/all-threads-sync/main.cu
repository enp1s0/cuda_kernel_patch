#include "hip/hip_runtime.h"
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <cuda_kernel_fusing.hpp>

constexpr std::size_t N = 1lu << 25;
constexpr std::size_t block_size = 1lu << 8;

int main() {
	cuda_kernel_fusing::kernel_constructor kernel_constructor(
			"float* const dst_ptr, const float* const src_ptr",
			"const unsigned tid, float& a",
			"const unsigned tid = threadIdx.x; float a = 1.0f;",
			"dst_ptr[tid] = a;"
			);

	kernel_constructor.debug_print_arguments();
	kernel_constructor.add_device_function(
			"device_func_0",
			R"(
{
	a *= src_ptr[tid];
}
)"
			);
	kernel_constructor.add_device_function(
			"device_func_1",
			R"(
{
	a /= src_ptr[tid];
}
)"
			);

	std::printf("# -- kernel code\n");
	const std::string kernel_code = kernel_constructor.generate_kernel_code({
				"device_func_0",
				"device_func_1",
				"device_func_0",
				"device_func_1",
				});
	std::cout << kernel_code << std::endl;

	hiprtcProgram program;
	hiprtcCreateProgram(&program,
			kernel_code.c_str(),
			"kernel.cu",
			0,
			NULL,
			NULL);
	const char *options[] = {
		"--gpu-architecture=compute_75",
	};
	hiprtcResult result = hiprtcCompileProgram(program, 1, options);
	size_t log_size;
	hiprtcGetProgramLogSize(program,&log_size);
	char *log = new char[log_size];
	hiprtcGetProgramLog(program,log);
	std::cout<<log<<std::endl;
	delete [] log;
	if(result != HIPRTC_SUCCESS){
		std::cerr<<"Compilation failed"<<std::endl;
		return 1;
	}

	// Get PTX
	std::size_t ptx_size;
	hiprtcGetCodeSize(program, &ptx_size);
	char *ptx = new char [ptx_size];
	hiprtcGetCode(program, ptx);
	hiprtcDestroyProgram(&program);

	// Create kernel image
	hipDevice_t cuDevice;
	hipCtx_t cuContext;
	hipModule_t cuModule;
	hipFunction_t cuFunction;
	hipInit(0);
	hipDeviceGet(&cuDevice, 0);
	hipCtxCreate(&cuContext, 0, cuDevice);
	hipModuleLoadDataEx(&cuModule, ptx, 0, 0, 0);
	hipModuleGetFunction(&cuFunction, cuModule, "cukf_main");
	delete [] ptx;

	// Launch
	float *dx, *dy;
	hipMalloc(&dx, sizeof(float) * N);
	hipMalloc(&dy, sizeof(float) * N);

	void *args[] = {&dy,&dx};
	hipModuleLaunchKernel(cuFunction,
			N / block_size,1,1,
			block_size,1,1,
			0, NULL,
			args, 0);
	hipCtxSynchronize();

	hipFree(dx);
	hipFree(dy);

	hipModuleUnload(cuModule);
	hipCtxDestroy(cuContext);
}
